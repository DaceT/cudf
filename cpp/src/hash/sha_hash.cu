#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/hashing.hpp>
#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/table/table_device_view.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>
#include <type_traits>

#include <thrust/for_each.h>
#include <thrust/iterator/constant_iterator.h>

namespace cudf {
namespace detail {

namespace {

// SHA supported leaf data type check
bool sha_type_check(data_type dt)
{
  return !is_chrono(dt) && (is_fixed_width(dt) || (dt.id() == type_id::STRING));
}

CUDA_DEVICE_CALLABLE uint32_t rotate_bits_left(uint32_t x, int8_t r)
{
  // Equivalent to (x << r) | (x >> (32 - r))
  return __funnelshift_l(x, x, r);
}

// Swap the endianness of a 32 bit value
CUDA_DEVICE_CALLABLE uint32_t swap_endian(uint32_t x)
{
  // The selector 0x0123 reverses the byte order
  return __byte_perm(x, 0, 0x0123);
}

// Swap the endianness of a 64 bit value
// There is no CUDA intrinsic for permuting bytes in 64 bit integers
CUDA_DEVICE_CALLABLE uint64_t swap_endian(uint64_t x)
{
  // Reverse the endianness of each 32 bit section
  uint32_t low_bits  = swap_endian(static_cast<uint32_t>(x));
  uint32_t high_bits = swap_endian(static_cast<uint32_t>(x >> 32));
  // Reassemble a 64 bit result, swapping the low bits and high bits
  return (static_cast<uint64_t>(low_bits) << 32) | (static_cast<uint64_t>(high_bits));
};

}  // namespace

template <typename Hasher, typename IntermediateType, typename WordType, uint32_t MessageChunkSize>
struct SHAHash {
  // Number of bytes processed in each hash step
  static constexpr auto message_chunk_size = MessageChunkSize;
  // Number of bytes used for the message length
  static constexpr auto message_length_size = 8;
  using sha_intermediate_data               = IntermediateType;
  using sha_word_type                       = WordType;

  /**
   * @brief Execute SHA on input data chunks.
   *
   * This accepts arbitrary data, handles it as bytes, and calls the hash step
   * when the buffer is filled up to message_chunk_size bytes.
   */
  template <typename TKey>
  void CUDA_DEVICE_CALLABLE process(TKey const& key, sha_intermediate_data* hash_state) const
  {
    uint32_t const len  = sizeof(TKey);
    uint8_t const* data = reinterpret_cast<uint8_t const*>(&key);
    hash_state->message_length += len;

    if (hash_state->buffer_length + len < message_chunk_size) {
      std::memcpy(hash_state->buffer + hash_state->buffer_length, data, len);
      hash_state->buffer_length += len;
    } else {
      uint32_t copylen = message_chunk_size - hash_state->buffer_length;

      std::memcpy(hash_state->buffer + hash_state->buffer_length, data, copylen);
      Hasher::hash_step(hash_state);

      while (len > message_chunk_size + copylen) {
        std::memcpy(hash_state->buffer, data + copylen, message_chunk_size);
        Hasher::hash_step(hash_state);
        copylen += message_chunk_size;
      }

      std::memcpy(hash_state->buffer, data + copylen, len - copylen);
      hash_state->buffer_length = len - copylen;
    }
  }

  /**
   * @brief Finalize SHA element processing.
   *
   * This method fills the remainder of the message buffer with zeros, appends
   * the message length (in another step of the hash, if needed), and performs
   * the final hash step.
   */
  void CUDA_DEVICE_CALLABLE finalize(sha_intermediate_data* hash_state, char* result_location)
  {
    // Message length in bits.
    uint64_t const message_length_in_bits = (static_cast<uint64_t>(hash_state->message_length))
                                            << 3;

    // Add a one bit flag to signal the end of the message
    constexpr uint8_t end_of_message = 0x80;
    // 1 byte for the end of the message flag
    constexpr int end_of_message_size = 1;

    thrust::fill_n(thrust::seq,
                   hash_state->buffer + hash_state->buffer_length,
                   end_of_message_size,
                   end_of_message);

    // SHA-512 uses a 128-bit message length instead of a 64-bit message length
    // but this code does not support messages with lengths exceeding UINT64_MAX
    // bits. We always pad the upper 64 bits with zeros.
    constexpr auto message_length_supported_size = sizeof(message_length_in_bits);

    if (hash_state->buffer_length + message_length_size + end_of_message_size <=
        message_chunk_size) {
      // Fill the remainder of the buffer with zeros up to the space reserved
      // for the message length. The message length fits in this hash step.
      thrust::fill_n(thrust::seq,
                     hash_state->buffer + hash_state->buffer_length + 1,
                     (message_chunk_size - message_length_supported_size - end_of_message_size -
                      hash_state->buffer_length),
                     0x00);
    } else {
      // Fill the remainder of the buffer with zeros. The message length doesn't
      // fit and will be processed in a subsequent hash step comprised of only
      // zeros followed by the message length.
      thrust::fill_n(thrust::seq,
                     hash_state->buffer + hash_state->buffer_length + end_of_message_size,
                     (message_chunk_size - hash_state->buffer_length),
                     0x00);
      Hasher::hash_step(hash_state);

      thrust::fill_n(
        thrust::seq, hash_state->buffer, message_chunk_size - message_length_size, 0x00);
    }

    // Convert the 64-bit message length from little-endian to big-endian.
    uint64_t const full_length_flipped = swap_endian(message_length_in_bits);
    std::memcpy(hash_state->buffer + message_chunk_size - message_length_supported_size,
                reinterpret_cast<uint8_t const*>(&full_length_flipped),
                message_length_supported_size);
    Hasher::hash_step(hash_state);

#pragma unroll
    for (int i = 0; i < 5; i++) {
      // Convert word representation from big-endian to little-endian.
      sha_word_type flipped = swap_endian(hash_state->hash_value[i]);
      if constexpr (std::is_same_v<sha_word_type, uint32_t>) {
        uint32ToLowercaseHexString(flipped, result_location + (8 * i));
      } else if constexpr (std::is_same_v<sha_word_type, uint32_t>) {
        uint32_t low_bits = static_cast<uint32_t>(flipped);
        uint32ToLowercaseHexString(low_bits, result_location + (16 * i));
        uint32_t high_bits = static_cast<uint32_t>(flipped >> 32);
        uint32ToLowercaseHexString(high_bits, result_location + (16 * i) + 8);
      } else {
        cudf_assert(false && "Unsupported SHA word type.");
      }
    }
  };

  template <typename T, typename std::enable_if_t<is_chrono<T>()>* = nullptr>
  void CUDA_DEVICE_CALLABLE operator()(column_device_view col,
                                       size_type row_index,
                                       sha_intermediate_data* hash_state) const
  {
    cudf_assert(false && "SHA Unsupported chrono type column");
  }

  template <
    typename T,
    typename std::enable_if_t<!is_fixed_width<T>() && !std::is_same_v<T, string_view>>* = nullptr>
  void CUDA_DEVICE_CALLABLE operator()(column_device_view col,
                                       size_type row_index,
                                       sha_intermediate_data* hash_state) const
  {
    cudf_assert(false && "SHA Unsupported non-fixed-width type column");
  }

  template <typename T, typename std::enable_if_t<is_floating_point<T>()>* = nullptr>
  void CUDA_DEVICE_CALLABLE operator()(column_device_view col,
                                       size_type row_index,
                                       sha_intermediate_data* hash_state) const
  {
    T const& key = col.element<T>(row_index);
    if (isnan(key)) {
      T nan = std::numeric_limits<T>::quiet_NaN();
      process(nan, hash_state);
    } else if (key == T{0.0}) {
      process(T{0.0}, hash_state);
    } else {
      process(key, hash_state);
    }
  }

  template <typename T,
            typename std::enable_if_t<is_fixed_width<T>() && !is_floating_point<T>() &&
                                      !is_chrono<T>()>* = nullptr>
  void CUDA_DEVICE_CALLABLE operator()(column_device_view col,
                                       size_type row_index,
                                       sha_intermediate_data* hash_state) const
  {
    process(col.element<T>(row_index), hash_state);
  }

  template <typename T, typename std::enable_if_t<std::is_same_v<T, string_view>>* = nullptr>
  void CUDA_DEVICE_CALLABLE operator()(column_device_view col,
                                       size_type row_index,
                                       sha_intermediate_data* hash_state) const
  {
    string_view key     = col.element<string_view>(row_index);
    uint32_t const len  = static_cast<uint32_t>(key.size_bytes());
    uint8_t const* data = reinterpret_cast<uint8_t const*>(key.data());
    hash_state->message_length += len;

    if (hash_state->buffer_length + len < message_chunk_size) {
      // If the buffer will not be filled by this data, we copy the new data into
      // the buffer but do not trigger a hash step yet.
      std::memcpy(hash_state->buffer + hash_state->buffer_length, data, len);
      hash_state->buffer_length += len;
    } else {
      // The buffer will be filled by this data. Copy a chunk of the data to fill
      // the buffer and trigger a hash step.
      uint32_t copylen = message_chunk_size - hash_state->buffer_length;
      std::memcpy(hash_state->buffer + hash_state->buffer_length, data, copylen);
      Hasher::hash_step(hash_state);

      // Take buffer-sized chunks of the data and do a hash step on each chunk.
      while (len > message_chunk_size + copylen) {
        std::memcpy(hash_state->buffer, data + copylen, message_chunk_size);
        Hasher::hash_step(hash_state);
        copylen += message_chunk_size;
      }

      // The remaining data chunk does not fill the buffer. We copy the data into
      // the buffer but do not trigger a hash step yet.
      std::memcpy(hash_state->buffer, data + copylen, len - copylen);
      hash_state->buffer_length = len - copylen;
    }
  }
};

struct SHA1Hash : SHAHash<SHA1Hash, sha1_intermediate_data, sha1_word_type, 64> {
  /**
   * @brief Core SHA-1 algorithm implementation. Processes a single 512-bit chunk,
   * updating the hash value so far. Does not zero out the buffer contents.
   */
  static void __device__ hash_step(sha_intermediate_data* hash_state)
  {
    sha_word_type A = hash_state->hash_value[0];
    sha_word_type B = hash_state->hash_value[1];
    sha_word_type C = hash_state->hash_value[2];
    sha_word_type D = hash_state->hash_value[3];
    sha_word_type E = hash_state->hash_value[4];

    sha_word_type words[80];

    // Word size in bytes
    constexpr auto word_size = sizeof(sha_word_type);

    // The 512-bit message buffer fills the first 16 words.
    for (int i = 0; i < 16; i++) {
      sha_word_type buffer_element_as_int;
      std::memcpy(&buffer_element_as_int, hash_state->buffer + (i * word_size), word_size);
      // Convert word representation from little-endian to big-endian.
      words[i] = swap_endian(buffer_element_as_int);
    }

    // The rest of the 80 words are generated from the first 16 words.
    for (int i = 16; i < 80; i++) {
      sha_word_type temp = words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16];
      words[i]           = rotate_bits_left(temp, 1);
    }

    for (int i = 0; i < 80; i++) {
      sha_word_type F;
      sha_word_type temp;
      sha_word_type k;
      switch (i / 20) {
        case 0:
          F = D ^ (B & (C ^ D));
          k = 0x5a827999;
          break;
        case 1:
          F = B ^ C ^ D;
          k = 0x6ed9eba1;
          break;
        case 2:
          F = (B & C) | (B & D) | (C & D);
          k = 0x8f1bbcdc;
          break;
        case 3:
          F = B ^ C ^ D;
          k = 0xca62c1d6;
          break;
      }
      temp = rotate_bits_left(A, 5) + F + E + k + words[i];
      E    = D;
      D    = C;
      C    = rotate_bits_left(B, 30);
      B    = A;
      A    = temp;
    }

    hash_state->hash_value[0] += A;
    hash_state->hash_value[1] += B;
    hash_state->hash_value[2] += C;
    hash_state->hash_value[3] += D;
    hash_state->hash_value[4] += E;

    hash_state->buffer_length = 0;
  }
};

std::unique_ptr<column> sha1_hash(table_view const& input,
                                  hipStream_t stream,
                                  rmm::mr::device_memory_resource* mr)
{
  if (input.num_columns() == 0 || input.num_rows() == 0) {
    // Return the SHA-1 hash of a zero-length input.
    const string_scalar string_160bit("da39a3ee5e6b4b0d3255bfef95601890afd80709");
    auto output = make_column_from_scalar(string_160bit, input.num_rows(), stream, mr);
    return output;
  }

  // Accepts string and fixed width columns.
  // TODO: Accept single layer list columns holding those types.
  CUDF_EXPECTS(
    std::all_of(input.begin(), input.end(), [](auto col) { return sha_type_check(col.type()); }),
    "SHA-1 unsupported column type");

  // Result column allocation and creation
  auto begin = thrust::make_constant_iterator(40);
  auto offsets_column =
    cudf::strings::detail::make_offsets_child_column(begin, begin + input.num_rows(), stream, mr);

  auto chars_column = strings::detail::create_chars_child_column(input.num_rows() * 40, stream, mr);
  auto chars_view   = chars_column->mutable_view();
  auto d_chars      = chars_view.data<char>();

  rmm::device_buffer null_mask{0, stream, mr};

  auto const device_input = table_device_view::create(input, stream);

  // Hash each row, hashing each element sequentially left to right
  thrust::for_each(rmm::exec_policy(stream),
                   thrust::make_counting_iterator(0),
                   thrust::make_counting_iterator(input.num_rows()),
                   [d_chars, device_input = *device_input] __device__(auto row_index) {
                     sha1_intermediate_data hash_state;
                     SHA1Hash hasher = SHA1Hash{};
                     for (int col_index = 0; col_index < device_input.num_columns(); col_index++) {
                       if (device_input.column(col_index).is_valid(row_index)) {
                         cudf::type_dispatcher<dispatch_storage_type>(
                           device_input.column(col_index).type(),
                           hasher,
                           device_input.column(col_index),
                           row_index,
                           &hash_state);
                       }
                     }
                     hasher.finalize(&hash_state, d_chars + (row_index * 40));
                   });

  return make_strings_column(
    input.num_rows(), std::move(offsets_column), std::move(chars_column), 0, std::move(null_mask));
}

std::unique_ptr<column> sha256_hash(table_view const& input,
                                    bool truncate_output,
                                    hipStream_t stream,
                                    rmm::mr::device_memory_resource* mr)
{
  return nullptr;
}

std::unique_ptr<column> sha512_hash(table_view const& input,
                                    bool truncate_output,
                                    hipStream_t stream,
                                    rmm::mr::device_memory_resource* mr)
{
  return nullptr;
}

}  // namespace detail
}  // namespace cudf
