#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/hashing.hpp>
#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/table/table_device_view.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/for_each.h>
#include <thrust/iterator/constant_iterator.h>

namespace cudf {
namespace detail {

namespace {

// SHA supported leaf data type check
bool sha_type_check(data_type dt)
{
  return !is_chrono(dt) && (is_fixed_width(dt) || (dt.id() == type_id::STRING));
}

}  // namespace

struct SHA1Hash {
  CUDA_DEVICE_CALLABLE uint32_t rotl32(uint32_t x, int8_t r) const
  {
    // Equivalent to (x << r) | (x >> (32 - r))
    return __funnelshift_l(x, x, r);
  }

  /**
   * @brief Core SHA-1 algorithm implementation. Processes a single 512-bit chunk,
   * updating the hash value so far. Does not zero out the buffer contents.
   */
  void __device__ hash_step(sha1_intermediate_data* hash_state) const
  {
    uint32_t A = hash_state->hash_value[0];
    uint32_t B = hash_state->hash_value[1];
    uint32_t C = hash_state->hash_value[2];
    uint32_t D = hash_state->hash_value[3];
    uint32_t E = hash_state->hash_value[4];

    uint32_t words[80];

    // The 512-bit message buffer fills the first 16 words.
    for (int i = 0; i < 16; i++) {
      uint32_t buffer_element_as_int;
      std::memcpy(&buffer_element_as_int, hash_state->buffer + (i * 4), 4);
      // Convert word representation from little-endian to big-endian.
      words[i] = __byte_perm(buffer_element_as_int, 0, 0x0123);
    }

    // The rest of the 80 words are generated from the first 16 words.
    for (int i = 16; i < 80; i++) {
      uint32_t temp = words[i - 3] ^ words[i - 8] ^ words[i - 14] ^ words[i - 16];
      words[i]      = rotl32(temp, 1);
    }

#pragma unroll
    for (int i = 0; i < 80; i++) {
      uint32_t F;
      uint32_t temp;
      uint32_t k;
      switch (i / 20) {
        case 0:
          F = D ^ (B & (C ^ D));
          k = 0x5a827999;
          break;
        case 1:
          F = B ^ C ^ D;
          k = 0x6ed9eba1;
          break;
        case 2:
          F = (B & C) | (B & D) | (C & D);
          k = 0x8f1bbcdc;
          break;
        case 3:
          F = B ^ C ^ D;
          k = 0xca62c1d6;
          break;
      }
      temp = rotl32(A, 5) + F + E + k + words[i];
      E    = D;
      D    = C;
      C    = rotl32(B, 30);
      B    = A;
      A    = temp;
    }

    hash_state->hash_value[0] += A;
    hash_state->hash_value[1] += B;
    hash_state->hash_value[2] += C;
    hash_state->hash_value[3] += D;
    hash_state->hash_value[4] += E;

    hash_state->buffer_length = 0;
  }

  /**
   * @brief Core SHA1 element processing function
   */
  template <typename TKey>
  void __device__ process(TKey const& key, sha1_intermediate_data* hash_state) const
  {
    uint32_t const len  = sizeof(TKey);
    uint8_t const* data = reinterpret_cast<uint8_t const*>(&key);
    hash_state->message_length += len;

    // 64 bytes are processed in each hash step
    constexpr int sha1_chunk_size = 64;
    if (hash_state->buffer_length + len < sha1_chunk_size) {
      std::memcpy(hash_state->buffer + hash_state->buffer_length, data, len);
      hash_state->buffer_length += len;
    } else {
      uint32_t copylen = sha1_chunk_size - hash_state->buffer_length;

      std::memcpy(hash_state->buffer + hash_state->buffer_length, data, copylen);
      hash_step(hash_state);

      while (len > sha1_chunk_size + copylen) {
        std::memcpy(hash_state->buffer, data + copylen, sha1_chunk_size);
        hash_step(hash_state);
        copylen += sha1_chunk_size;
      }

      std::memcpy(hash_state->buffer, data + copylen, len - copylen);
      hash_state->buffer_length = len - copylen;
    }
  }

  void __device__ finalize(sha1_intermediate_data* hash_state, char* result_location) const
  {
    // Message length in bits
    auto const full_length = (static_cast<uint64_t>(hash_state->message_length)) << 3;

    // Add a one bit flag to signal the end of the message
    thrust::fill_n(thrust::seq, hash_state->buffer + hash_state->buffer_length, 1, 0x80);

    // 64 bytes are processed in each hash step
    constexpr int sha1_chunk_size = 64;
    // 8 bytes for the total message length, appended to the end of the last chunk processed
    constexpr int message_length_size = 8;
    // 1 byte for the end of the message flag
    constexpr int end_of_message_size = 1;
    if (hash_state->buffer_length + message_length_size + end_of_message_size <= sha1_chunk_size) {
      // Fill the remainder of the buffer with zeros
      thrust::fill_n(
        thrust::seq,
        hash_state->buffer + hash_state->buffer_length + 1,
        (sha1_chunk_size - message_length_size - end_of_message_size - hash_state->buffer_length),
        0x00);
    } else {
      thrust::fill_n(thrust::seq,
                     hash_state->buffer + hash_state->buffer_length + 1,
                     (sha1_chunk_size - hash_state->buffer_length),
                     0x00);
      hash_step(hash_state);

      thrust::fill_n(thrust::seq, hash_state->buffer, sha1_chunk_size - message_length_size, 0x00);
    }

    // Convert the 64-bit message length from little-endian to big-endian.
    // There is currently no CUDA intrinsic for permuting bytes in 64 bit integers.
    auto uint64_swap_endian = [](uint64_t x) -> uint64_t {
      // Reverse the endianness of each 32 bit section
      uint32_t low_bits  = __byte_perm(x, 0, 0x123);
      uint32_t high_bits = __byte_perm(x >> 32, 0, 0x123);
      // Reassemble a 64 bit result
      uint64_t y = (static_cast<uint64_t>(low_bits) << 32) | (static_cast<uint64_t>(high_bits));
      return y;
    };

    auto const full_length_flipped = uint64_swap_endian(full_length);
    std::memcpy(hash_state->buffer + sha1_chunk_size - message_length_size,
                reinterpret_cast<uint8_t const*>(&full_length_flipped),
                message_length_size);
    hash_step(hash_state);
    // std::memcpy(hash_state->hash_value, hash_state->buffer, 160);

#pragma unroll
    for (int i = 0; i < 5; ++i) {
      // Convert word representation from big-endian to little-endian.
      uint32_t flipped = __byte_perm(hash_state->hash_value[i], 0, 0x0123);
      uint32ToLowercaseHexString(flipped, result_location + (8 * i));
    }
  }

  template <typename T, typename std::enable_if_t<is_chrono<T>()>* = nullptr>
  void __device__ operator()(column_device_view col,
                             size_type row_index,
                             sha1_intermediate_data* hash_state) const
  {
    cudf_assert(false && "SHA-1 Unsupported chrono type column");
  }

  template <typename T, typename std::enable_if_t<!is_fixed_width<T>()>* = nullptr>
  void __device__ operator()(column_device_view col,
                             size_type row_index,
                             sha1_intermediate_data* hash_state) const
  {
    cudf_assert(false && "SHA-1 Unsupported non-fixed-width type column");
  }

  template <typename T, typename std::enable_if_t<is_floating_point<T>()>* = nullptr>
  void __device__ operator()(column_device_view col,
                             size_type row_index,
                             sha1_intermediate_data* hash_state) const
  {
    T const& key = col.element<T>(row_index);
    if (isnan(key)) {
      T nan = std::numeric_limits<T>::quiet_NaN();
      process(nan, hash_state);
    } else if (key == T{0.0}) {
      process(T{0.0}, hash_state);
    } else {
      process(key, hash_state);
    }
  }

  template <typename T,
            typename std::enable_if_t<is_fixed_width<T>() && !is_floating_point<T>() &&
                                      !is_chrono<T>()>* = nullptr>
  void CUDA_DEVICE_CALLABLE operator()(column_device_view col,
                                       size_type row_index,
                                       sha1_intermediate_data* hash_state) const
  {
    process(col.element<T>(row_index), hash_state);
  }
};

template <>
void CUDA_DEVICE_CALLABLE SHA1Hash::operator()<string_view>(
  column_device_view col, size_type row_index, sha1_intermediate_data* hash_state) const
{
  string_view key     = col.element<string_view>(row_index);
  uint32_t const len  = static_cast<uint32_t>(key.size_bytes());
  uint8_t const* data = reinterpret_cast<uint8_t const*>(key.data());
  hash_state->message_length += len;

  // 64 bytes are processed in each hash step
  constexpr int sha1_chunk_size = 64;
  if (hash_state->buffer_length + len < sha1_chunk_size) {
    // If the buffer will not be filled by this data, we copy the new data into
    // the buffer but do not trigger a hash step yet.
    std::memcpy(hash_state->buffer + hash_state->buffer_length, data, len);
    hash_state->buffer_length += len;
  } else {
    // The buffer will be filled by this data. Copy a chunk of the data to fill
    // the buffer and trigger a hash step.
    uint32_t copylen = sha1_chunk_size - hash_state->buffer_length;
    std::memcpy(hash_state->buffer + hash_state->buffer_length, data, copylen);
    hash_step(hash_state);

    // Take buffer-sized chunks of the data and do a hash step on each chunk.
    while (len > sha1_chunk_size + copylen) {
      std::memcpy(hash_state->buffer, data + copylen, sha1_chunk_size);
      hash_step(hash_state);
      copylen += sha1_chunk_size;
    }

    // The remaining data chunk does not fill the buffer. We copy the data into
    // the buffer but do not trigger a hash step yet.
    std::memcpy(hash_state->buffer, data + copylen, len - copylen);
    hash_state->buffer_length = len - copylen;
  }
}

std::unique_ptr<column> sha1_hash(table_view const& input,
                                  hipStream_t stream,
                                  rmm::mr::device_memory_resource* mr)
{
  if (input.num_columns() == 0 || input.num_rows() == 0) {
    // Return the SHA-1 hash of a zero-length input.
    const string_scalar string_160bit("da39a3ee5e6b4b0d3255bfef95601890afd80709");
    auto output = make_column_from_scalar(string_160bit, input.num_rows(), stream, mr);
    return output;
  }

  // Accepts string and fixed width columns.
  // TODO: Accept single layer list columns holding those types.
  CUDF_EXPECTS(
    std::all_of(input.begin(), input.end(), [](auto col) { return sha_type_check(col.type()); }),
    "SHA-1 unsupported column type");

  // Result column allocation and creation
  auto begin = thrust::make_constant_iterator(40);
  auto offsets_column =
    cudf::strings::detail::make_offsets_child_column(begin, begin + input.num_rows(), stream, mr);

  auto chars_column = strings::detail::create_chars_child_column(input.num_rows() * 40, stream, mr);
  auto chars_view   = chars_column->mutable_view();
  auto d_chars      = chars_view.data<char>();

  rmm::device_buffer null_mask{0, stream, mr};

  auto const device_input = table_device_view::create(input, stream);

  // Hash each row, hashing each element sequentially left to right
  thrust::for_each(rmm::exec_policy(stream),
                   thrust::make_counting_iterator(0),
                   thrust::make_counting_iterator(input.num_rows()),
                   [d_chars, device_input = *device_input] __device__(auto row_index) {
                     sha1_intermediate_data hash_state;
                     SHA1Hash hasher = SHA1Hash{};
                     for (int col_index = 0; col_index < device_input.num_columns(); col_index++) {
                       if (device_input.column(col_index).is_valid(row_index)) {
                         cudf::type_dispatcher<dispatch_storage_type>(
                           device_input.column(col_index).type(),
                           hasher,
                           device_input.column(col_index),
                           row_index,
                           &hash_state);
                       }
                     }
                     hasher.finalize(&hash_state, d_chars + (row_index * 40));
                   });

  return make_strings_column(
    input.num_rows(), std::move(offsets_column), std::move(chars_column), 0, std::move(null_mask));
}

std::unique_ptr<column> sha256_hash(table_view const& input,
                                    bool truncate_output,
                                    hipStream_t stream,
                                    rmm::mr::device_memory_resource* mr)
{
  return nullptr;
}

std::unique_ptr<column> sha512_hash(table_view const& input,
                                    bool truncate_output,
                                    hipStream_t stream,
                                    rmm::mr::device_memory_resource* mr)
{
  return nullptr;
}

}  // namespace detail
}  // namespace cudf
