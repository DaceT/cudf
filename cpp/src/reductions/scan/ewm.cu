#include "hip/hip_runtime.h"
#include "scan.cuh"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/reduction.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/scan.h>

#include <type_traits>


namespace cudf {
namespace detail {

class blelloch_functor {
public:
 __device__ thrust::pair<double, double> operator()(thrust::pair<double, double> ci,
                                                    thrust::pair<double, double> cj)
 {
   double ci0 = thrust::get<0>(ci);
   double ci1 = thrust::get<1>(ci);
   double cj0 = thrust::get<0>(cj);
   double cj1 = thrust::get<1>(cj);
   return thrust::pair<double, double>(ci0 * cj0, ci1 * cj0 + cj1);
 }
};

/**
* @brief Solve a recurrence relation using a blelloch scan
* The second elements of the pairs will contain the result
*/
void compute_recurrence(rmm::device_uvector<thrust::pair<double, double>>& input,
                       rmm::cuda_stream_view stream)
{
 blelloch_functor op;
 thrust::inclusive_scan(rmm::exec_policy(stream), input.begin(), input.end(), input.begin(), op);
}

/**
* @brief Return an array whose values y_i are the number of null entries
* in between the last valid entry of the input and the current index.
* Example: {1, NULL, 3, 4, NULL, NULL, 7}
        -> {0, 0     1, 0, 0,    1,    2}
*/
rmm::device_uvector<double> null_roll_up(column_view const& input, rmm::cuda_stream_view stream)
{
 rmm::device_uvector<double> output(input.size(), stream, rmm::mr::get_current_device_resource());

 auto device_view = *column_device_view::create(input);
 auto valid_it    = cudf::detail::make_validity_iterator(device_view);

 // TODO - not sure why two iterators produce a different result
 // Invert the null iterator
 thrust::transform(rmm::exec_policy(stream),
                   valid_it,
                   valid_it + input.size(),
                   output.begin(),
                   [=] __host__ __device__(bool valid) -> bool { return 1 - valid; });

 // 0, 1, 0, 1, 1, 0 -> 0, 0, 1, 0, 0, 2
 thrust::inclusive_scan_by_key(
   rmm::exec_policy(stream), output.begin(), output.end() - 1, output.begin(), output.begin() + 1);

 return output;
}

/**
* @brief modify the source pairs that eventually yield the numerator
* and denoninator to account for nan values. Pairs at nan indicies
* become the identity operator (1, 0). The first pair after a nan
* value or sequence of nan values has its first element multiplied by
* N factors of beta, where N is the number of preceeding NaNs.
*/
void pair_beta_adjust(column_view const& input,
                     rmm::device_uvector<thrust::pair<double, double>>& pairs,
                     rmm::cuda_stream_view stream)
{
 rmm::device_uvector<double> nullcnt = null_roll_up(input, stream);

 auto device_view = *column_device_view::create(input);
 auto valid_it    = cudf::detail::make_validity_iterator(device_view);
 thrust::transform(
   rmm::exec_policy(stream),
   valid_it,
   valid_it + input.size(),
   pairs.begin(),
   pairs.begin(),
   [=] __host__ __device__(bool valid,
                           thrust::pair<double, double> pair) -> thrust::pair<double, double> {
     if (!valid) {
       return thrust::pair<double, double>(1.0, 0.0);
     } else {
       return pair;
     }
   });

 valid_it           = cudf::detail::make_validity_iterator(device_view);
 auto valid_and_exp = thrust::make_zip_iterator(thrust::make_tuple(valid_it, nullcnt.begin()));

 thrust::transform(
   rmm::exec_policy(stream),
   valid_and_exp,
   valid_and_exp + input.size(),
   pairs.begin(),
   pairs.begin(),
   [=] __host__ __device__(thrust::tuple<bool, int> valid_and_exp,
                           thrust::pair<double, double> pair) -> thrust::pair<double, double> {
     bool valid = thrust::get<0>(valid_and_exp);
     int exp    = thrust::get<1>(valid_and_exp);
     if (valid & (exp != 0)) {
       double beta  = thrust::get<0>(pair);
       double scale = thrust::get<1>(pair);
       return thrust::pair<double, double>(beta * (pow(beta, exp)), scale);
     } else {
       return pair;
     }
   });
}

rmm::device_uvector<double> compute_ewma_adjust(column_view const& input,
                                               double beta,
                                               rmm::cuda_stream_view stream,
                                               rmm::mr::device_memory_resource* mr)
{
 rmm::device_uvector<double> output(input.size(), stream, mr);
 rmm::device_uvector<thrust::pair<double, double>> pairs(input.size(), stream, mr);

 // Numerator
 // Fill with pairs
 thrust::transform(rmm::exec_policy(stream),
                   input.begin<double>(),
                   input.end<double>(),
                   pairs.begin(),
                   [=] __host__ __device__(double input) -> thrust::pair<double, double> {
                     return thrust::pair<double, double>(beta, input);
                   });

 if (input.has_nulls()) { pair_beta_adjust(input, pairs, stream); }

 compute_recurrence(pairs, stream);

 // copy the second elements to the output for now
 thrust::transform(rmm::exec_policy(stream),
                   pairs.begin(),
                   pairs.end(),
                   output.begin(),
                   [=] __host__ __device__(thrust::pair<double, double> pair) -> double {
                     return thrust::get<1>(pair);
                   });

 // Denominator
 // Fill with pairs
 thrust::fill(
   rmm::exec_policy(stream), pairs.begin(), pairs.end(), thrust::pair<double, double>(beta, 1.0));

 if (input.has_nulls()) { pair_beta_adjust(input, pairs, stream); }
 compute_recurrence(pairs, stream);

 thrust::transform(
   rmm::exec_policy(stream),
   pairs.begin(),
   pairs.end(),
   output.begin(),
   output.begin(),
   [=] __host__ __device__(thrust::pair<double, double> pair, double numerator) -> double {
     return numerator / thrust::get<1>(pair);
   });
 return output;
}

rmm::device_uvector<double> compute_ewma_noadjust(column_view const& input,
                                                 double beta,
                                                 rmm::cuda_stream_view stream,
                                                 rmm::mr::device_memory_resource* mr)
{
 rmm::device_uvector<double> output(input.size(), stream, mr);
 rmm::device_uvector<thrust::pair<double, double>> pairs(input.size(), stream, mr);

 thrust::transform(rmm::exec_policy(stream),
                   input.begin<double>(),
                   input.end<double>(),
                   pairs.begin(),
                   [=] __host__ __device__(double input) -> thrust::pair<double, double> {
                     return thrust::pair<double, double>(beta, (1.0 - beta) * input);
                   });

 // TODO: the first pair is WRONG using the above. Reset just that pair

 thrust::transform(rmm::exec_policy(stream),
                   input.begin<double>(),
                   input.begin<double>() + 1,
                   pairs.begin(),
                   [=] __host__ __device__(double input) -> thrust::pair<double, double> {
                     return thrust::pair<double, double>(beta, input);
                   });

 if (input.has_nulls()) {
   /*
   In this case, a denominator actually has to be computed. The formula is
   y_{i+1} - (1 - alpha)x_{i-1} + alpha x_i, but really there is a "denominator"
   which is the sum of the weights: alpha + (1 - alpha) == 1. If a null is
   encountered, that means that the "previous" value is downweighted by a
   factor (for each missing value). For example this would y_2 be for one NULL:
   data = {x_0, NULL, x_1},
   y_2 = (1 - alpha)**2 x_0 + alpha * x_2 / (alpha + (1-alpha)**2)

   As such, the pairs must be updated before summing like the adjusted case,
   but we also have to compute normalization factors

   */
   pair_beta_adjust(input, pairs, stream);

   rmm::device_uvector<double> nullcnt = null_roll_up(input, stream);

   thrust::transform(rmm::exec_policy(stream),
                     nullcnt.begin(),
                     nullcnt.end(),
                     nullcnt.begin(),
                     [=] __host__ __device__(double exponent) -> double {
                       // ex: 2 -> alpha + (1  - alpha)**2
                       if (exponent != 0) {
                         return (1.0 - beta) + pow(beta, exponent + 1);
                       } else {
                         return exponent;
                       }
                     });

   auto device_view = *column_device_view::create(input);
   auto valid_it    = detail::make_validity_iterator(device_view);
   auto null_and_null_count =
     thrust::make_zip_iterator(thrust::make_tuple(valid_it, nullcnt.begin()));
   thrust::transform(
     rmm::exec_policy(stream),
     null_and_null_count,
     null_and_null_count + input.size(),
     pairs.begin(),
     pairs.begin(),
     [=] __host__ __device__(thrust::tuple<bool, double> null_and_null_count,
                             thrust::pair<double, double> pair) -> thrust::pair<double, double> {
       bool is_valid = thrust::get<0>(null_and_null_count);
       double factor = thrust::get<1>(null_and_null_count);

       double ci = thrust::get<0>(pair);
       double cj = thrust::get<1>(pair);

       if (is_valid and (factor != 0.0)) {
         return {ci / factor, cj / factor};
       } else {
         return {ci, cj};
       }
     });
 }
 compute_recurrence(pairs, stream);
 // copy the second elements to the output for now
 thrust::transform(rmm::exec_policy(stream),
                   pairs.begin(),
                   pairs.end(),
                   output.begin(),
                   [=] __host__ __device__(thrust::pair<double, double> pair) -> double {
                     return thrust::get<1>(pair);
                   });
 return output;
}

std::unique_ptr<column> ewma(column_view const& input,
                            double com,
                            bool adjust,
                            rmm::cuda_stream_view stream,
                            rmm::mr::device_memory_resource* mr)
{
 CUDF_EXPECTS(cudf::is_floating_point(input.type()), "Column must be floating point type");

 double beta = 1.0 - (1.0 / (com + 1.0));

 rmm::device_uvector<double> data(input.size(), stream, mr);
 if (adjust) {
   data = compute_ewma_adjust(input, beta, stream, mr);
 } else {
   data = compute_ewma_noadjust(input, beta, stream, mr);
 }
 auto col = std::make_unique<column>(
   cudf::data_type{cudf::type_id::FLOAT64}, input.size(), std::move(data.release()));
 return col;
}

void print_device_uvector(rmm::device_uvector<double> const& input, rmm::cuda_stream_view stream) {
 thrust::device_vector<double> input_device(input.size());
 thrust::copy(rmm::exec_policy(stream), input.begin(), input.end(), input_device.begin());
 thrust::host_vector<double> input_host = input_device;
 std::cout << std::endl;
 for (size_t i = 0; i < input_host.size(); i++) {
   std::cout << input_host[i] << " ";
 }
 std::cout << std::endl;
}


std::unique_ptr<column> ewm(column_view const& input,
                           std::unique_ptr<aggregation> const& agg,
                           rmm::cuda_stream_view stream,
                           rmm::mr::device_memory_resource* mr)
{
 switch (agg->kind) {
   case aggregation::EWMA: {
     double com  = (dynamic_cast<ewma_aggregation*>(agg.get()))->com;
     bool adjust = (dynamic_cast<ewma_aggregation*>(agg.get()))->adjust;
     return ewma(input, com, adjust, stream, mr);
   }
   default: CUDF_FAIL("Unsupported aggregation operator for scan");
 }
}

}  // namespace detail
}  // namespace cudf
