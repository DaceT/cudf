#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "scan.cuh"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/reduction.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/scan.h>

namespace cudf {
namespace detail {

// logical-and scan of the null mask of the input view
rmm::device_buffer mask_scan(column_view const& input_view,
                             scan_type inclusive,
                             rmm::cuda_stream_view stream,
                             rmm::mr::device_memory_resource* mr)
{
  rmm::device_buffer mask =
    detail::create_null_mask(input_view.size(), mask_state::UNINITIALIZED, stream, mr);
  auto d_input   = column_device_view::create(input_view, stream);
  auto valid_itr = detail::make_validity_iterator(*d_input);

  auto first_null_position = [&] {
    size_type const first_null = thrust::find_if_not(rmm::exec_policy(stream),
                                                     valid_itr,
                                                     valid_itr + input_view.size(),
                                                     thrust::identity<bool>{}) -
                                 valid_itr;
    size_type const exclusive_offset = (inclusive == scan_type::EXCLUSIVE) ? 1 : 0;
    return std::min(input_view.size(), first_null + exclusive_offset);
  }();

  set_null_mask(static_cast<bitmask_type*>(mask.data()), 0, first_null_position, true, stream);
  set_null_mask(
    static_cast<bitmask_type*>(mask.data()), first_null_position, input_view.size(), false, stream);
  return mask;
}

namespace {

/**
 * @brief Min/Max inclusive scan operator
 *
 * This operator will accept index values, check them and then
 * run the `Op` operation on the individual element objects.
 * The returned result is the appropriate index value.
 *
 * This was specifically created to workaround a thrust issue
 * https://github.com/NVIDIA/thrust/issues/1479
 * where invalid values are passed to the operator.
 */
template <typename Element, typename Op>
struct min_max_scan_operator {
  column_device_view const col;      ///< strings column device view
  Element const null_replacement{};  ///< value used when element is null
  bool const has_nulls;              ///< true if col has null elements

  min_max_scan_operator(column_device_view const& col, bool has_nulls = true)
    : col{col}, null_replacement{Op::template identity<Element>()}, has_nulls{has_nulls}
  {
    // verify validity bitmask is non-null, otherwise, is_null_nocheck() will crash
    if (has_nulls) CUDF_EXPECTS(col.nullable(), "column with nulls must have a validity bitmask");
  }

  CUDA_DEVICE_CALLABLE
  size_type operator()(size_type lhs, size_type rhs) const
  {
    // thrust::inclusive_scan may pass us garbage values so we need to protect ourselves;
    // in these cases the return value does not matter since the result is not used
    if (lhs < 0 || rhs < 0 || lhs >= col.size() || rhs >= col.size()) return 0;
    Element d_lhs =
      has_nulls && col.is_null_nocheck(lhs) ? null_replacement : col.element<Element>(lhs);
    Element d_rhs =
      has_nulls && col.is_null_nocheck(rhs) ? null_replacement : col.element<Element>(rhs);
    return Op{}(d_lhs, d_rhs) == d_lhs ? lhs : rhs;
  }
};

template <typename Op, typename T>
struct scan_functor {
  static std::unique_ptr<column> invoke(column_view const& input_view,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
  {
    auto output_column = detail::allocate_like(
      input_view, input_view.size(), mask_allocation_policy::NEVER, stream, mr);
    mutable_column_view result = output_column->mutable_view();

    auto d_input = column_device_view::create(input_view, stream);
    auto const begin =
      make_null_replacement_iterator(*d_input, Op::template identity<T>(), input_view.has_nulls());
    thrust::inclusive_scan(
      rmm::exec_policy(stream), begin, begin + input_view.size(), result.data<T>(), Op{});

    CHECK_CUDA(stream.value());
    return output_column;
  }
};

template <typename Op>
struct scan_functor<Op, cudf::string_view> {
  static std::unique_ptr<column> invoke(column_view const& input_view,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
  {
    auto d_input = column_device_view::create(input_view, stream);

    // build indices of the scan operation results
    rmm::device_uvector<size_type> result(input_view.size(), stream);
    thrust::inclusive_scan(
      rmm::exec_policy(stream),
      thrust::counting_iterator<size_type>(0),
      thrust::counting_iterator<size_type>(input_view.size()),
      result.begin(),
      min_max_scan_operator<cudf::string_view, Op>{*d_input, input_view.has_nulls()});

    // call gather using the indices to build the output column
    auto result_table = cudf::detail::gather(cudf::table_view({input_view}),
                                             result,
                                             out_of_bounds_policy::DONT_CHECK,
                                             negative_index_policy::NOT_ALLOWED,
                                             stream,
                                             mr);
    return std::move(result_table->release().front());
  }
};

/**
 * @brief Dispatcher for running a Scan operation on an input column
 *
 * @tparam Op device binary operator
 */
template <typename Op>
struct scan_dispatcher {
 private:
  template <typename T>
  static constexpr bool is_min_max_supported()
  {
    return cudf::is_relationally_comparable<T, T>() && !cudf::is_dictionary<T>() &&
           (std::is_same_v<Op, DeviceMin> || std::is_same_v<Op, DeviceMax>);
  }

  template <typename T>
  static constexpr bool is_supported()
  {
    return std::is_arithmetic_v<T> || is_min_max_supported<T>();
  }

 public:
  /**
   * @brief Creates a new column from the input column by applying the scan operation
   *
   * @param input Input column view
   * @param null_handling How null row entries are to be processed
   * @param stream CUDA stream used for device memory operations and kernel launches.
   * @param mr Device memory resource used to allocate the returned column's device memory
   * @return
   *
   * @tparam T type of input column
   */
  template <typename T, typename std::enable_if_t<is_supported<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& input,
                                     null_policy,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
  {
    return scan_functor<Op, T>::invoke(input, stream, mr);
  }

  template <typename T, typename... Args>
  std::enable_if_t<!is_supported<T>(), std::unique_ptr<column>> operator()(Args&&...)
  {
    CUDF_FAIL("Unsupported type for inclusive scan operation");
  }
};

}  // namespace


class blelloch_functor {
 public:
  __device__ thrust::pair<double, double> operator()(thrust::pair<double, double> ci,
                                                     thrust::pair<double, double> cj)
  {
    double ci0 = thrust::get<0>(ci);
    double ci1 = thrust::get<1>(ci);
    double cj0 = thrust::get<0>(cj);
    double cj1 = thrust::get<1>(cj);
    return thrust::pair<double, double>(ci0 * cj0, ci1 * cj0 + cj1);
  }
};

/**
 * @brief Solve a recurrence relation using a blelloch scan
 * The second elements of the pairs will contain the result
 */
void compute_recurrence(rmm::device_uvector<thrust::pair<double, double>>& input,
                        rmm::cuda_stream_view stream)
{
  blelloch_functor op;
  thrust::inclusive_scan(rmm::exec_policy(stream), input.begin(), input.end(), input.begin(), op);
}

/**
 * @brief Return an array whose values y_i are the number of null entries
 * in between the last valid entry of the input and the current index.
 * Example: {1, NULL, 3, 4, NULL, NULL, 7}
         -> {0, 0     1, 0, 0,    1,    2}
 */
rmm::device_uvector<double> null_roll_up(column_view const& input, rmm::cuda_stream_view stream)
{
  rmm::device_uvector<double> output(input.size(), stream, rmm::mr::get_current_device_resource());

  auto device_view = *column_device_view::create(input);
  auto valid_it    = cudf::detail::make_validity_iterator(device_view);

  // TODO - not sure why two iterators produce a different result
  // Invert the null iterator
  thrust::transform(rmm::exec_policy(stream),
                    valid_it,
                    valid_it + input.size(),
                    output.begin(),
                    [=] __host__ __device__(bool valid) -> bool { return 1 - valid; });

  // 0, 1, 0, 1, 1, 0 -> 0, 0, 1, 0, 0, 2
  thrust::inclusive_scan_by_key(
    rmm::exec_policy(stream), output.begin(), output.end() - 1, output.begin(), output.begin() + 1);

  return output;
}

/**
 * @brief modify the source pairs that eventually yield the numerator
 * and denoninator to account for nan values. Pairs at nan indicies
 * become the identity operator (1, 0). The first pair after a nan
 * value or sequence of nan values has its first element multiplied by
 * N factors of beta, where N is the number of preceeding NaNs.
 */
void pair_beta_adjust(column_view const& input,
                      rmm::device_uvector<thrust::pair<double, double>>& pairs,
                      rmm::cuda_stream_view stream)
{
  rmm::device_uvector<double> nullcnt = null_roll_up(input, stream);

  auto device_view = *column_device_view::create(input);
  auto valid_it    = cudf::detail::make_validity_iterator(device_view);
  thrust::transform(
    rmm::exec_policy(stream),
    valid_it,
    valid_it + input.size(),
    pairs.begin(),
    pairs.begin(),
    [=] __host__ __device__(bool valid,
                            thrust::pair<double, double> pair) -> thrust::pair<double, double> {
      if (!valid) {
        return thrust::pair<double, double>(1.0, 0.0);
      } else {
        return pair;
      }
    });

  valid_it           = cudf::detail::make_validity_iterator(device_view);
  auto valid_and_exp = thrust::make_zip_iterator(thrust::make_tuple(valid_it, nullcnt.begin()));

  thrust::transform(
    rmm::exec_policy(stream),
    valid_and_exp,
    valid_and_exp + input.size(),
    pairs.begin(),
    pairs.begin(),
    [=] __host__ __device__(thrust::tuple<bool, int> valid_and_exp,
                            thrust::pair<double, double> pair) -> thrust::pair<double, double> {
      bool valid = thrust::get<0>(valid_and_exp);
      int exp    = thrust::get<1>(valid_and_exp);
      if (valid & (exp != 0)) {
        double beta  = thrust::get<0>(pair);
        double scale = thrust::get<1>(pair);
        return thrust::pair<double, double>(beta * (pow(beta, exp)), scale);
      } else {
        return pair;
      }
    });
}

rmm::device_uvector<double> compute_ewma_adjust(column_view const& input,
                                                double beta,
                                                rmm::cuda_stream_view stream,
                                                rmm::mr::device_memory_resource* mr)
{
  rmm::device_uvector<double> output(input.size(), stream, mr);
  rmm::device_uvector<thrust::pair<double, double>> pairs(input.size(), stream, mr);

  // Numerator
  // Fill with pairs
  thrust::transform(rmm::exec_policy(stream),
                    input.begin<double>(),
                    input.end<double>(),
                    pairs.begin(),
                    [=] __host__ __device__(double input) -> thrust::pair<double, double> {
                      return thrust::pair<double, double>(beta, input);
                    });

  if (input.has_nulls()) { pair_beta_adjust(input, pairs, stream); }

  compute_recurrence(pairs, stream);

  // copy the second elements to the output for now
  thrust::transform(rmm::exec_policy(stream),
                    pairs.begin(),
                    pairs.end(),
                    output.begin(),
                    [=] __host__ __device__(thrust::pair<double, double> pair) -> double {
                      return thrust::get<1>(pair);
                    });

  // Denominator
  // Fill with pairs
  thrust::fill(
    rmm::exec_policy(stream), pairs.begin(), pairs.end(), thrust::pair<double, double>(beta, 1.0));

  if (input.has_nulls()) { pair_beta_adjust(input, pairs, stream); }
  compute_recurrence(pairs, stream);

  thrust::transform(
    rmm::exec_policy(stream),
    pairs.begin(),
    pairs.end(),
    output.begin(),
    output.begin(),
    [=] __host__ __device__(thrust::pair<double, double> pair, double numerator) -> double {
      return numerator / thrust::get<1>(pair);
    });
  return output;
}

rmm::device_uvector<double> compute_ewma_noadjust(column_view const& input,
                                                  double beta,
                                                  rmm::cuda_stream_view stream,
                                                  rmm::mr::device_memory_resource* mr)
{
  rmm::device_uvector<double> output(input.size(), stream, mr);
  rmm::device_uvector<thrust::pair<double, double>> pairs(input.size(), stream, mr);

  thrust::transform(rmm::exec_policy(stream),
                    input.begin<double>(),
                    input.end<double>(),
                    pairs.begin(),
                    [=] __host__ __device__(double input) -> thrust::pair<double, double> {
                      return thrust::pair<double, double>(beta, (1.0 - beta) * input);
                    });

  // TODO: the first pair is WRONG using the above. Reset just that pair

  thrust::transform(rmm::exec_policy(stream),
                    input.begin<double>(),
                    input.begin<double>() + 1,
                    pairs.begin(),
                    [=] __host__ __device__(double input) -> thrust::pair<double, double> {
                      return thrust::pair<double, double>(beta, input);
                    });

  if (input.has_nulls()) {
    /*
    In this case, a denominator actually has to be computed. The formula is
    y_{i+1} - (1 - alpha)x_{i-1} + alpha x_i, but really there is a "denominator"
    which is the sum of the weights: alpha + (1 - alpha) == 1. If a null is
    encountered, that means that the "previous" value is downweighted by a
    factor (for each missing value). For example this would y_2 be for one NULL:
    data = {x_0, NULL, x_1},
    y_2 = (1 - alpha)**2 x_0 + alpha * x_2 / (alpha + (1-alpha)**2)

    As such, the pairs must be updated before summing like the adjusted case,
    but we also have to compute normalization factors

    */
    pair_beta_adjust(input, pairs, stream);

    rmm::device_uvector<double> nullcnt = null_roll_up(input, stream);

    thrust::transform(rmm::exec_policy(stream),
                      nullcnt.begin(),
                      nullcnt.end(),
                      nullcnt.begin(),
                      [=] __host__ __device__(double exponent) -> double {
                        // ex: 2 -> alpha + (1  - alpha)**2
                        if (exponent != 0) {
                          return (1.0 - beta) + pow(beta, exponent + 1);
                        } else {
                          return exponent;
                        }
                      });

    auto device_view = *column_device_view::create(input);
    auto valid_it    = detail::make_validity_iterator(device_view);
    auto null_and_null_count =
      thrust::make_zip_iterator(thrust::make_tuple(valid_it, nullcnt.begin()));
    thrust::transform(
      rmm::exec_policy(stream),
      null_and_null_count,
      null_and_null_count + input.size(),
      pairs.begin(),
      pairs.begin(),
      [=] __host__ __device__(thrust::tuple<bool, double> null_and_null_count,
                              thrust::pair<double, double> pair) -> thrust::pair<double, double> {
        bool is_valid = thrust::get<0>(null_and_null_count);
        double factor = thrust::get<1>(null_and_null_count);

        double ci = thrust::get<0>(pair);
        double cj = thrust::get<1>(pair);

        if (is_valid and (factor != 0.0)) {
          return {ci / factor, cj / factor};
        } else {
          return {ci, cj};
        }
      });
  }
  compute_recurrence(pairs, stream);
  // copy the second elements to the output for now
  thrust::transform(rmm::exec_policy(stream),
                    pairs.begin(),
                    pairs.end(),
                    output.begin(),
                    [=] __host__ __device__(thrust::pair<double, double> pair) -> double {
                      return thrust::get<1>(pair);
                    });
  return output;
}

std::unique_ptr<column> ewma(column_view const& input,
                             double com,
                             bool adjust,
                             rmm::cuda_stream_view stream,
                             rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(cudf::is_floating_point(input.type()), "Column must be floating point type");

  double beta = 1.0 - (1.0 / (com + 1.0));

  rmm::device_uvector<double> data(input.size(), stream, mr);
  if (adjust) {
    data = compute_ewma_adjust(input, beta, stream, mr);
  } else {
    data = compute_ewma_noadjust(input, beta, stream, mr);
  }
  auto col = std::make_unique<column>(
    cudf::data_type{cudf::type_id::FLOAT64}, input.size(), std::move(data.release()));
  return col;
}

void print_device_uvector(rmm::device_uvector<double> const& input, rmm::cuda_stream_view stream) {
  thrust::device_vector<double> input_device(input.size());
  thrust::copy(rmm::exec_policy(stream), input.begin(), input.end(), input_device.begin());
  thrust::host_vector<double> input_host = input_device;
  std::cout << std::endl;
  for (size_t i = 0; i < input_host.size(); i++) {
    std::cout << input_host[i] << " ";
  }
  std::cout << std::endl;
}

/**
 * @brief Compute exponentially weighted moving variance.
 * The simplest definition for EWMVAR is defined is
 * EWMVAR[i] = EWMA[xi**2] - EWMA[xi]**2. Those EWMA are
 * themselves calculated with adjust=true/false, leading
 * to two types of EWMVAR calculations. From there, EWMVAR
 * may be biased or unbiased, leading to four cases. Finally,
 * nulls can either be present or not, which requires special
 * handling in every case. This leads to eight possibilities.
 */
std::unique_ptr<column> ewmvar(column_view const& input,
                               double com,
                               bool adjust,
                               bool bias,
                               rmm::cuda_stream_view stream,
                               rmm::mr::device_memory_resource* mr)
{

  if (!adjust and bias and input.has_nulls()) {
    // unadjusted, biased result with no nulls
  }


  // get xi**2
  std::unique_ptr<column> xi_sqr = make_fixed_width_column(
    cudf::data_type{cudf::type_id::FLOAT64}, input.size(), copy_bitmask(input));
  mutable_column_view xi_sqr_d = xi_sqr->mutable_view();
  thrust::transform(rmm::exec_policy(stream),
                    input.begin<double>(),
                    input.end<double>(),
                    xi_sqr_d.begin<double>(),
                    [=] __host__ __device__(double input) -> double { return input * input; });

  // get EWMA[xi**2]
  std::unique_ptr<column> ewma_xi_sqr = ewma((*xi_sqr).view(), com, adjust, stream, mr);

  // get EWMA[xi]
  std::unique_ptr<column> ewma_xi = ewma(input, com, adjust, stream, mr);

  // reuse the memory from computing xi_sqr to write the output
  thrust::transform(
    rmm::exec_policy(stream),
    ewma_xi.get()[0].view().begin<double>(),
    ewma_xi.get()[0].view().end<double>(),
    ewma_xi_sqr.get()[0].view().begin<double>(),
    ewma_xi.get()[0].mutable_view().begin<double>(),
    [=] __host__ __device__(double x, double xsqrd) -> double { return xsqrd - x * x; });


  if (!bias) {
    /*
    We're going to need to compute SUM(w_i ** 2) for each index, this is another pair summation.
    

    
    
    */
    double beta = 1.0 - (1.0 / (com + 1.0));

    if (adjust) {
      // allocate pairs

      rmm::device_uvector<double> bias(input.size(), stream, mr);

      rmm::device_uvector<thrust::pair<double, double>> pairs(input.size(), stream, mr);
      thrust::fill(rmm::exec_policy(stream), pairs.begin(), pairs.end(), thrust::pair<double, double>(beta*beta, 1));
      compute_recurrence(pairs, stream);
      
      thrust::transform(rmm::exec_policy(stream),
                        pairs.begin(),
                        pairs.end(),
                        bias.begin(),
                        [=] __host__ __device__ (thrust::pair<double, double> input) -> double {
                          return thrust::get<1>(input);
                        }
      );
      thrust::fill(rmm::exec_policy(stream), pairs.begin(), pairs.end(), thrust::pair<double, double>(beta, 1));
      compute_recurrence(pairs, stream);

      thrust::transform(rmm::exec_policy(stream),
                        pairs.begin(),
                        pairs.end(),
                        bias.begin(),
                        bias.begin(),
                        [=] __host__ __device__ (thrust::pair<double, double> pair, double wisqr) -> double {
                          double wi = thrust::get<1>(pair);
                          
                          return (wi*wi) / ((wi*wi) - wisqr);
                        }
      );
      print_device_uvector(bias, stream);
      thrust::transform(rmm::exec_policy(stream),
                        bias.begin(),
                        bias.end(),
                        ewma_xi.get()[0].mutable_view().begin<double>(),
                        ewma_xi.get()[0].mutable_view().begin<double>(),
                        [=] __host__ __device__ (double bias, double input) -> double {
                          return bias * input;
                        }
      
      );

    } else {
      if (input.has_nulls()) {
        double beta = 1.0 - (1.0 / (com + 1.0));
        double alpha = 1.0 - beta;
        auto d_input   = column_device_view::create(input, stream);
        auto valid_itr = detail::make_validity_iterator(*d_input);

        rmm::device_uvector<double> nullcnt = null_roll_up(input, stream);
        thrust::transform(rmm::exec_policy(stream), 
                          valid_itr, 
                          valid_itr + input.size(), 
                          nullcnt.begin(), 
                          nullcnt.begin(),
                          [=] __host__ __device__ (bool valid, double num_nulls) -> double {
                            double other_weight;
                            double wi;
                            double wisqr;
                            if (valid) {
                              if (num_nulls != 0.0) {
                                other_weight = beta;
                              } else {
                                other_weight = pow(beta, num_nulls + 1);
                              }
                              wi = alpha + other_weight;
                              wisqr = alpha * alpha + other_weight*other_weight;
                              return wi * wi / (wi * wi - wisqr);
                            } else {
                              return 1.0;
                            }
                          }
                        );

        thrust::transform(rmm::exec_policy(stream),
                          nullcnt.begin(),
                          nullcnt.end(),
                          ewma_xi.get()[0].mutable_view().begin<double>(),
                          ewma_xi.get()[0].mutable_view().begin<double>(),
                          thrust::multiplies<double>()
        );
      } else {
        // the weights will always be (beta, 1 - beta)
        double wi = beta + (1.0 - beta);
        double wisqr = beta * beta + (1.0 - beta) * (1.0 - beta);
        double const bias = wi * wi / (wi * wi - wisqr);
        auto mutable_view = ewma_xi.get()[0].mutable_view();

        rmm::device_uvector<double> bias_vec(input.size(), stream, mr);
        thrust::fill(rmm::exec_policy(stream), bias_vec.begin(), bias_vec.end(), bias);

        thrust::transform(rmm::exec_policy(stream),
                          mutable_view.begin<double>(),
                          mutable_view.end<double>(),
                          bias_vec.begin(),
                          mutable_view.begin<double>(),
                          thrust::multiplies<double>()
        );
      }
    }
  }
  // return means;
  return ewma_xi;
}

std::unique_ptr<column> ewmstd(column_view const& input,
                               double com,
                               bool adjust,
                               bool bias,
                               rmm::cuda_stream_view stream,
                               rmm::mr::device_memory_resource* mr)
{
  std::unique_ptr<column> var = ewmvar(input, com, adjust, bias, stream, mr);
  auto var_view               = var.get()[0].mutable_view();

  // write into the same memory
  thrust::transform(rmm::exec_policy(stream),
                    var_view.begin<double>(),
                    var_view.end<double>(),
                    var_view.begin<double>(),
                    [=] __host__ __device__(double input) -> double { return sqrt(input); });

  return var;
}

std::unique_ptr<column> ewm(column_view const& input,
                            std::unique_ptr<aggregation> const& agg,
                            rmm::cuda_stream_view stream,
                            rmm::mr::device_memory_resource* mr)
{
  switch (agg->kind) {
    case aggregation::EWMA: {
      double com  = (dynamic_cast<ewma_aggregation*>(agg.get()))->com;
      bool adjust = (dynamic_cast<ewma_aggregation*>(agg.get()))->adjust;
      return ewma(input, com, adjust, stream, mr);
    }
    case aggregation::EWMVAR: {
      double com  = (dynamic_cast<ewmvar_aggregation*>(agg.get()))->com;
      bool adjust = (dynamic_cast<ewmvar_aggregation*>(agg.get()))->adjust;
      bool bias = (dynamic_cast<ewmvar_aggregation*>(agg.get()))->bias;
      return ewmvar(input, com, adjust, bias, stream, mr);
    }
    case aggregation::EWMSTD: {
      double com  = (dynamic_cast<ewmstd_aggregation*>(agg.get()))->com;
      bool adjust = (dynamic_cast<ewmstd_aggregation*>(agg.get()))->adjust;
      bool bias = (dynamic_cast<ewmstd_aggregation*>(agg.get()))->bias;
      return ewmstd(input, com, adjust, bias, stream, mr);
    }
    default: CUDF_FAIL("Unsupported aggregation operator for scan");
  }
}

std::unique_ptr<column> scan_inclusive(
  column_view const& input,
  std::unique_ptr<aggregation> const& agg,
  null_policy null_handling,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  auto output = scan_agg_dispatch<scan_dispatcher>(input, agg, null_handling, stream, mr);

  if (agg->kind == aggregation::RANK || agg->kind == aggregation::DENSE_RANK ||
      agg->kind == aggregation::EWMA || agg->kind == aggregation::EWMVAR ||
      agg->kind == aggregation::EWMSTD) {
    return output;
  } else if (null_handling == null_policy::EXCLUDE) {
    output->set_null_mask(detail::copy_bitmask(input, stream, mr), input.null_count());
  } else if (input.nullable()) {
    output->set_null_mask(mask_scan(input, scan_type::INCLUSIVE, stream, mr), UNKNOWN_NULL_COUNT);
  }

  return output;
}
}  // namespace detail
}  // namespace cudf
