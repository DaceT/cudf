#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <nvtext/minhash.hpp>

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/hashing.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/sequence.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/device_atomics.cuh>
#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/strings/string_view.cuh>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>

#include <limits>

namespace nvtext {
namespace detail {
namespace {

struct minhash_fn {
  cudf::column_device_view d_strings;
  cudf::device_span<cudf::hash_value_type const> seeds;
  cudf::size_type width;
  cudf::hash_value_type* d_hashes;

  __device__ void operator()(cudf::size_type idx)
  {
    auto const str_idx  = idx / cudf::detail::warp_size;
    auto const lane_idx = idx % cudf::detail::warp_size;

    if (d_strings.is_null(str_idx)) { return; }
    auto const d_str = d_strings.element<cudf::string_view>(str_idx);
    for (auto seed_idx = 0; seed_idx < static_cast<cudf::size_type>(seeds.size()); ++seed_idx) {
      auto const output_idx = str_idx * seeds.size() + seed_idx;
      d_hashes[output_idx]  = d_str.empty() ? 0 : std::numeric_limits<cudf::hash_value_type>::max();
    }
    auto const begin = d_str.begin() + lane_idx;
    auto const end   = (d_str.length() <= width) ? d_str.end() : d_str.end() - (width - 1);

    for (auto itr = begin; itr < end; itr += cudf::detail::warp_size) {
      auto const offset = itr.byte_offset();
      auto const ss =
        cudf::string_view(d_str.data() + offset, (itr + width).byte_offset() - offset);

      for (auto seed_idx = 0; seed_idx < static_cast<cudf::size_type>(seeds.size()); ++seed_idx) {
        auto const output_idx = str_idx * seeds.size() + seed_idx;

        auto const seed   = seeds[seed_idx];
        auto const hasher = cudf::detail::MurmurHash3_32<cudf::string_view>{seed};

        auto const hvalue = hasher(ss);
        // cudf::detail::hash_combine(seed, hasher(ss)); <-- matches cudf::hash() result
        atomicMin(d_hashes + output_idx, hvalue);
      }
    }
  }
};

}  // namespace

std::unique_ptr<cudf::column> minhash(cudf::strings_column_view const& input,
                                      cudf::device_span<cudf::hash_value_type const> seeds,
                                      cudf::size_type width,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(width > 1, "Parameter width should be an integer value of 2 or greater");

  auto output_type = cudf::data_type{cudf::type_to_id<cudf::hash_value_type>()};
  if (input.is_empty()) { return cudf::make_empty_column(output_type); }

  auto const d_strings = cudf::column_device_view::create(input.parent(), stream);

  auto hashes   = cudf::make_numeric_column(output_type,
                                          input.size() * static_cast<cudf::size_type>(seeds.size()),
                                          cudf::mask_state::UNALLOCATED,
                                          stream,
                                          mr);
  auto d_hashes = hashes->mutable_view().data<cudf::hash_value_type>();

  thrust::for_each_n(rmm::exec_policy(stream),
                     thrust::counting_iterator<cudf::size_type>(0),
                     input.size() * cudf::detail::warp_size,
                     minhash_fn{*d_strings, seeds, width, d_hashes});

  if (seeds.size() == 1) {
    hashes->set_null_mask(cudf::detail::copy_bitmask(input.parent(), stream, mr),
                          input.null_count());
    return hashes;
  }
  hashes->set_null_count(0);

  auto offsets = cudf::detail::sequence(
    input.size() + 1,
    cudf::numeric_scalar<cudf::size_type>(0),
    cudf::numeric_scalar<cudf::size_type>(static_cast<cudf::size_type>(seeds.size())),
    stream,
    mr);
  return make_lists_column(input.size(),
                           std::move(offsets),
                           std::move(hashes),
                           input.null_count(),
                           cudf::detail::copy_bitmask(input.parent(), stream, mr),
                           stream,
                           mr);
}

}  // namespace detail

std::unique_ptr<cudf::column> minhash(cudf::strings_column_view const& input,
                                      cudf::numeric_scalar<cudf::hash_value_type> seed,
                                      cudf::size_type width,
                                      rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  auto seeds = cudf::device_span<cudf::hash_value_type const>{seed.data(), 1};
  return detail::minhash(input, seeds, width, cudf::get_default_stream(), mr);
}

std::unique_ptr<cudf::column> minhash(cudf::strings_column_view const& input,
                                      cudf::device_span<cudf::hash_value_type const> seeds,
                                      cudf::size_type width,
                                      rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::minhash(input, seeds, width, cudf::get_default_stream(), mr);
}

}  // namespace nvtext
